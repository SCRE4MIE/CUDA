#include "hip/hip_runtime.h"
/*
TESTED ON: RTX 4070 TI AND AMD RYZEN 7 7800X3D
GPU WALL TIME: 155.86s
CPU WALL TIME: 927.14s
==================================================
Multiplication of two 2D matrices.

Each thread corresponds to one operation: multiplying an element 
from matrix A by matrix B and then summing the result to the element 
of the resulting matrix.
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void kernel(int*arr_a, int*arr_b, int*arr_c, int m, int n, int k, long long int num_of_operations){

    long long int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
    long long int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if (threadId < num_of_operations){
        int x_a = threadId / (m*n);
        int y_a = threadId % m;

        int x_b = threadId % m;
        int y_b = (threadId / m ) % n;

        int x_c = threadId / (m*n);
        int y_c = (threadId / m ) % n;

        int multiple = arr_a[x_a * m + y_a] * arr_b[x_b * n + y_b];
        atomicAdd(&(arr_c[x_c * n + y_c]), (int)multiple);
    }
}

void displayMatrix(int*array, int m, int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            printf("%d, ", array[i*m+j]);
        }
        printf("\n");
    }
    printf("\n");
}


void mult_matrix_cpu(int *matrix1, int *matrix2, int *result, int rows1, int cols1, int cols2) {
    for (int i = 0; i < rows1; i++) {
        for (int j = 0; j < cols2; j++) {
            int sum = 0;
            for (int k = 0; k < cols1; k++) {
                sum += matrix1[i * cols1 + k] * matrix2[k * cols2 + j];
            }
            result[i * cols2 + j] = sum;
        }
    }
}

int main(void){

    const int m = 10000;
    const int n = 6000;
    const int k = 10000;

    // host pointers
    int *h_arr_a, *h_arr_b, *h_gpu_result, *h_cpu_result;
    // allocate memory for host pointers
    h_arr_a = (int*)malloc(m*n*sizeof(int));
    h_arr_b = (int*)malloc(n*k*sizeof(int));
    h_gpu_result = (int*)malloc(n*n*sizeof(int));
    h_cpu_result = (int*)malloc(n*n*sizeof(int));

    //initialize host pointers
    for(int i=0;i<m*n;i++){
        h_arr_a[i] = 1;
    }
    for(int i=0;i<n*k;i++){
        h_arr_b[i] = 1;
    }
    
    // Start measuring gpu time-----------------
    clock_t begin = clock();

    //device pointers
    hipError_t error;
    int *d_arr_a, *d_arr_b, *d_arr_c;
    error = hipMalloc((void**)&d_arr_a, m*n*sizeof(int));
    if(error != hipSuccess){
        fprintf(stderr, "Error: %s \n", hipGetErrorString(error));
    }
    error = hipMalloc((void**)&d_arr_b, n*k*sizeof(int));
     if(error != hipSuccess){
        fprintf(stderr, "Error: %s \n", hipGetErrorString(error));
    }
    error = hipMalloc((void**)&d_arr_c, n*n*sizeof(int));
     if(error != hipSuccess){
        fprintf(stderr, "Error: %s \n", hipGetErrorString(error));
    }
    // copy array a and b to a GPU memory
    hipMemcpy(d_arr_a, h_arr_a, m*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr_b, h_arr_b, n*k*sizeof(int), hipMemcpyHostToDevice);

    // kernel launch
    long long int num_of_operations = (long long int)n * n * m; // operation is a multiple a element and b element and += to c result index (num of threads)
    
    dim3 block(32,32,1);
    dim3 grid(18750,18750,1);

    kernel<<<grid, block>>>(d_arr_a, d_arr_b, d_arr_c, m, n, k, num_of_operations);
    hipMemcpy(h_gpu_result, d_arr_c, n*n*sizeof(int), hipMemcpyDeviceToHost);

    // Stop measuring gpu time-----------------
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("GPU time: %4.6fs\n\n", time_spent);

    // cpu implementation----------------------------------------

    // Start measuring cpu time-----------------
    begin = clock();
    mult_matrix_cpu(h_arr_a, h_arr_b, h_cpu_result, n, m, n);
    // Stop measuring cpu time-----------------
    end = clock();
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("CPU time: %4.6fs\n\n", time_spent);

    // display matrix a, b, gpu result and cpu_result

    // displayMatrix(h_arr_a, m, n);
    // displayMatrix(h_arr_b, n, k);
    // printf("GPU result:\n");
    // displayMatrix(h_gpu_result, n, n);
    // printf("CPU result:\n");
    // displayMatrix(h_cpu_result, n, n);

    // check cpu vs gpu:

    bool checkpoint = true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(h_cpu_result[i *n + j] != h_gpu_result[i *n + j]){
                checkpoint = false;
                printf("%d != %d\n", h_cpu_result[i *n + j], h_gpu_result[i *n + j]);
            }
        }
    }
    if(checkpoint == false){
        printf("Error, gpu result != cpu result!");
    }
    
    // free memory
    hipFree(d_arr_a);
    hipFree(d_arr_b);
    hipFree(d_arr_c);
    free(h_gpu_result);
    free(h_cpu_result);
    free(h_arr_a);
    free(h_arr_b);

    return 0;
}